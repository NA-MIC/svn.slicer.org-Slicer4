#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

extern "C" {
#include "CUDA_renderAlgo.h"
}

// includes, project
#include <cutil.h>
//#include "vtkType.h"
// includes, kernels

#define BLOCK_DIM2D 16// this must be set to 4 or more
#define SQR(X) ((X) * (X) )

__device__ unsigned char interpolate(float posX, float posY, float posZ,
				     unsigned char val1,
				     unsigned char val2,
				     unsigned char val3,
				     unsigned char val4,
				     unsigned char val5,
				     unsigned char val6,
				     unsigned char val7,
				     unsigned char val8){
  float revX=1-posX;
  float revY=1-posY;
  float revZ=1-posZ;
  
  return ((unsigned char) (revX*revY*revZ* val1+
			   revX*revY*posZ* val2+
			   revX*posY*revZ* val3+
			   revX*posY*posZ* val4+
			   posX*revY*revZ* val5+
			   posX*revY*posZ* val6+
			   posX*posY*revZ* val7+
			   posX*posY*posZ* val8)
	  );
}

template <typename T>
__global__ void CUDAkernel_renderAlgo_doIntegrationRender(
							  const cudaRendererInformation renInfo,
							  const cudaVolumeInformation volInfo
							  )
{
  int xIndex = blockDim.x *blockIdx.x + threadIdx.x;
  int yIndex = blockDim.y *blockIdx.y + threadIdx.y;

  __shared__ float2 s_minmaxTrace[BLOCK_DIM2D*BLOCK_DIM2D]; //starting and ending step of ray tracing 
  __shared__ float s_rayMap[BLOCK_DIM2D*BLOCK_DIM2D*6]; //ray map: position and orientation of ray after translation and rotation transformation
  __shared__ float s_dsize[3]; //display size (x, y, dummy)
  __shared__ float s_vsize[3]; //voxel dimension
  __shared__ float s_size[3]; //3D data size
  __shared__ float s_minmax[6]; //region of interest of 3D data (minX, maxX, minY, maxY, minZ, maxZ)
  __shared__ float s_remainingOpacity[BLOCK_DIM2D*BLOCK_DIM2D]; //integration value of alpha
  __shared__ unsigned char s_outputVal[BLOCK_DIM2D*BLOCK_DIM2D*3]; //output value
  __shared__ float s_zBuffer[BLOCK_DIM2D*BLOCK_DIM2D]; // z buffer

  float test;

  int tempacc=threadIdx.x+threadIdx.y*BLOCK_DIM2D; //index in grid

  __syncthreads();

  //copying variables into shared memory

  if(tempacc <3){ 
    s_dsize[xIndex%2]=renInfo.Resolution[xIndex%2];
    s_vsize[xIndex%3]=volInfo.VoxelSize[xIndex%3];
    s_size[xIndex%3]=volInfo.VolumeSize[xIndex%3];
  }else if(tempacc < 9){ 
    s_minmax[xIndex%6]=volInfo.MinMaxValue[xIndex%6];
  }

  __syncthreads();

  int outindex=xIndex+yIndex*s_dsize[0]; // index of result image

  //initialization of variables in shared memory

  s_remainingOpacity[tempacc]=1.0;
  s_outputVal[tempacc*3]=0;
  s_outputVal[tempacc*3+1]=0;
  s_outputVal[tempacc*3+2]=0;
  s_zBuffer[tempacc]=renInfo.ZBuffer[outindex]; //renInfo.ClippingRange[0] + renInfo.ZBuffer[outindex] * (renInfo.ClippingRange[1] - renInfo.ClippingRange[0]);
    
  __syncthreads();

  // lens map for perspective projection

  /*
    camera model start here
  */
  
  s_rayMap[tempacc*6]=renInfo.CameraPos[0] ;//+ s_size[0]*s_vsize[0]/2.0f;
  s_rayMap[tempacc*6+1]=renInfo.CameraPos[1] ;//+ s_size[1]*s_vsize[1]/2.0f;
  s_rayMap[tempacc*6+2]=renInfo.CameraPos[2] ;//+ s_size[2]*s_vsize[2]/2.0f;
  
  float vecX, vecY, vecZ;

  vecX=(renInfo.TargetPos[0]-renInfo.CameraPos[0]);
  vecY=(renInfo.TargetPos[1]-renInfo.CameraPos[1]);
  vecZ=(renInfo.TargetPos[2]-renInfo.CameraPos[2]);

  float temp= 1.0f/sqrt(vecX*vecX+vecY*vecY+vecZ*vecZ);
  vecX*=temp;
  vecY*=temp;
  vecZ*=temp;

  float verX, verY, verZ;
  float horX, horY, horZ;
  
  float dot = renInfo.ViewUp[0]*vecX+renInfo.ViewUp[1]*vecY+renInfo.ViewUp[2]*vecZ;

  verX=renInfo.ViewUp[0]-dot*vecX;
  verY=renInfo.ViewUp[1]-dot*vecY;
  verZ=renInfo.ViewUp[2]-dot*vecZ;

  temp= 1.0f/sqrt(verX*verX+verY*verY+verZ*verZ);
  verX*=temp;
  verY*=temp;
  verZ*=temp;

  horX=verY*vecZ-verZ*vecY;
  horY=verZ*vecX-verX*vecZ;
  horZ=verX*vecY-verY*vecX;

  float posHor=(xIndex-s_dsize[0]*0.5)/s_dsize[0]*0.27;
  float posVer=(yIndex-s_dsize[1]*0.5)/s_dsize[0]*0.27;
  
  s_rayMap[tempacc*6+3]=(vecX+posHor*horX+posVer*verX);
  s_rayMap[tempacc*6+4]=(vecY+posHor*horY+posVer*verY);
  s_rayMap[tempacc*6+5]=(vecZ+posHor*horZ+posVer*verZ);

  /*
    camera model end here
  */
 
  //initialize variables for calculating starting and ending point of ray tracing

  s_minmaxTrace[tempacc].x=-100000.0f;
  s_minmaxTrace[tempacc].y=100000.0f;

  __syncthreads();
  
  //normalize ray vector

  float getmax = fabs(s_rayMap[tempacc*6+3]/s_vsize[0]);
  if(fabs(s_rayMap[tempacc*6+4]/s_vsize[1])>getmax) getmax = fabs(s_rayMap[tempacc*6+4]/s_vsize[1]);
  if(fabs(s_rayMap[tempacc*6+5]/s_vsize[2])>getmax) getmax = fabs(s_rayMap[tempacc*6+5]/s_vsize[2]);

  if(getmax!=0){
    float temp= 1.0f/getmax;
    s_rayMap[tempacc*6+3]*=temp;
    s_rayMap[tempacc*6+4]*=temp;
    s_rayMap[tempacc*6+5]*=temp;
  }

  float stepSize=sqrt(s_rayMap[tempacc*6+3]*s_rayMap[tempacc*6+3]+s_rayMap[tempacc*6+4]*s_rayMap[tempacc*6+4]+s_rayMap[tempacc*6+5]*s_rayMap[tempacc*6+5]);
  
  __syncthreads();

  //calculating starting and ending point of ray tracing

 if(s_rayMap[tempacc*6+3] > 1.0e-3){
    s_minmaxTrace[tempacc].y = ( ((s_minmax[1]-2)*s_vsize[0]-s_rayMap[tempacc*6])/s_rayMap[tempacc*6+3] );
    s_minmaxTrace[tempacc].x = ( ((s_minmax[0]+2)*s_vsize[0]-s_rayMap[tempacc*6])/s_rayMap[tempacc*6+3] );
  }
  else if(s_rayMap[tempacc*6+3] < -1.0e-3){
    s_minmaxTrace[tempacc].x = ( ((s_minmax[1]-2)*s_vsize[0]-s_rayMap[tempacc*6])/s_rayMap[tempacc*6+3] );
    s_minmaxTrace[tempacc].y = ( ((s_minmax[0]+2)*s_vsize[0]-s_rayMap[tempacc*6])/s_rayMap[tempacc*6+3] );
  }
  
  if(s_rayMap[tempacc*6+4] > 1.0e-3){
    test = ( ((s_minmax[3]-2)*s_vsize[1]-s_rayMap[tempacc*6+1])/s_rayMap[tempacc*6+4] );
    if( test < s_minmaxTrace[tempacc].y){
      s_minmaxTrace[tempacc].y = test;
    }
    test = ( ((s_minmax[2]+2)*s_vsize[1]-s_rayMap[tempacc*6+1])/s_rayMap[tempacc*6+4] );
    if( test > s_minmaxTrace[tempacc].x){
      s_minmaxTrace[tempacc].x = test;
    }
  }
  else if(s_rayMap[tempacc*6+4] < -1.0e-3){
    test = ( ((s_minmax[3]-2)*s_vsize[1]-s_rayMap[tempacc*6+1])/s_rayMap[tempacc*6+4] );
    if( test > s_minmaxTrace[tempacc].x){
      s_minmaxTrace[tempacc].x = test;
    }
    test = ( ((s_minmax[2]+2)*s_vsize[1]-s_rayMap[tempacc*6+1])/s_rayMap[tempacc*6+4] );
    if( test < s_minmaxTrace[tempacc].y){
      s_minmaxTrace[tempacc].y = test;
    }
  }
  

  if(s_rayMap[tempacc*6+5] > 1.0e-3){
    test = ( ((s_minmax[5]-2)*s_vsize[2]-s_rayMap[tempacc*6+2])/s_rayMap[tempacc*6+5] );
    if( test < s_minmaxTrace[tempacc].y){
      s_minmaxTrace[tempacc].y = test;
    }
    test = ( ((s_minmax[4]+2)*s_vsize[2]-s_rayMap[tempacc*6+2])/s_rayMap[tempacc*6+5] );
    if( test > s_minmaxTrace[tempacc].x){
      s_minmaxTrace[tempacc].x = test;
    }
  }
  else if(s_rayMap[tempacc*6+5] < -1.0e-3){
    test = ( ((s_minmax[5]-2)*s_vsize[2]-s_rayMap[tempacc*6+2])/s_rayMap[tempacc*6+5] );
    if( test > s_minmaxTrace[tempacc].x){
      s_minmaxTrace[tempacc].x = test;
    }
    test = ( ((s_minmax[4]+2)*s_vsize[2]-s_rayMap[tempacc*6+2])/s_rayMap[tempacc*6+5] );
    if( test < s_minmaxTrace[tempacc].y){
      s_minmaxTrace[tempacc].y = test;
    }
  }
  __syncthreads();

  //ray tracing start from here

  float tempx,tempy,tempz; // variables to store current position
  int pos=0; //current step distance from camera

  //float temp; //temporary variable to store data during calculation
  float alpha; //alpha value of current voxel
  float initialZBuffer=s_zBuffer[tempacc]; //initial zBuffer from input

  //perform ray tracing until integration of alpha value reach threshold 
  
  while((s_minmaxTrace[tempacc].y-s_minmaxTrace[tempacc].x)>=pos){
    
    //calculate current position in ray tracing

    tempx = ( s_rayMap[tempacc*6+0]+((int)s_minmaxTrace[tempacc].x+pos)*s_rayMap[tempacc*6+3]);
    tempy = ( s_rayMap[tempacc*6+1]+((int)s_minmaxTrace[tempacc].x+pos)*s_rayMap[tempacc*6+4]);
    tempz = ( s_rayMap[tempacc*6+2]+((int)s_minmaxTrace[tempacc].x+pos)*s_rayMap[tempacc*6+5]);
    
    tempx /= s_vsize[0];
    tempy /= s_vsize[1];
    tempz /= s_vsize[2];
    

    if(tempx >= s_minmax[0] && tempx <= s_minmax[1] && tempy >= s_minmax[2] && tempy <= s_minmax[3] && tempz >= s_minmax[4] && tempz <= s_minmax[5] && pos+s_minmaxTrace[tempacc].x >= -500 /*renInfo.ClippingRange[0]*/){ // if current position is in ROI

      if((pos+s_minmaxTrace[tempacc].x)*stepSize < initialZBuffer){ //check whether current position is in front of z buffer wall

	temp=((T*)volInfo.SourceData)[(int)(__float2int_rn(tempz)*s_size[0]*s_size[1]+__float2int_rn(tempy)*s_size[0]+__float2int_rn(tempx))];
	
	/*interpolation start here*/
	/*
	float posX=tempx-(int)tempx;
	float posY=tempy-(int)tempy;
	float posZ=tempz-(int)tempz;

	temp=interpolate(posX, posY, posZ,
			 ((T*)volInfo.SourceData)[(int)((int)(tempz)*s_size[0]*s_size[1]+(int)(tempy)*s_size[0]+(int)(tempx))],
			 ((T*)volInfo.SourceData)[(int)((int)(tempz+1)*s_size[0]*s_size[1]+(int)(tempy)*s_size[0]+(int)(tempx))],
			 ((T*)volInfo.SourceData)[(int)((int)(tempz)*s_size[0]*s_size[1]+(int)(tempy+1)*s_size[0]+(int)(tempx))],
			 ((T*)volInfo.SourceData)[(int)((int)(tempz+1)*s_size[0]*s_size[1]+(int)(tempy+1)*s_size[0]+(int)(tempx))],
			 ((T*)volInfo.SourceData)[(int)((int)(tempz)*s_size[0]*s_size[1]+(int)(tempy)*s_size[0]+(int)(tempx+1))],
			 ((T*)volInfo.SourceData)[(int)((int)(tempz+1)*s_size[0]*s_size[1]+(int)(tempy)*s_size[0]+(int)(tempx+1))],
			 ((T*)volInfo.SourceData)[(int)((int)(tempz)*s_size[0]*s_size[1]+(int)(tempy+1)*s_size[0]+(int)(tempx+1))],
			 ((T*)volInfo.SourceData)[(int)((int)(tempz+1)*s_size[0]*s_size[1]+(int)(tempy+1)*s_size[0]+(int)(tempx+1))]);
	*/

	/*interpolation end here*/

	if( temp >=(T)volInfo.MinThreshold && temp <= (T)volInfo.MaxThreshold){ 

	  alpha=volInfo.AlphaTransferFunction[(int)temp];
	  
	  if(s_zBuffer[tempacc] > (pos+s_minmaxTrace[tempacc].x)*stepSize){
	    s_zBuffer[tempacc]=(pos+s_minmaxTrace[tempacc].x)*stepSize;
	  }
	  if(s_remainingOpacity[tempacc]>0.02){ // check if remaining opacity has reached threshold(0.02)
	    s_outputVal[tempacc*3]+=s_remainingOpacity[tempacc]*alpha*volInfo.ColorTransferFunction[(int)temp*3]*256.0;
	    s_outputVal[tempacc*3+1]+=s_remainingOpacity[tempacc]*alpha*volInfo.ColorTransferFunction[(int)temp*3+1]*256.0;
	    s_outputVal[tempacc*3+2]+=s_remainingOpacity[tempacc]*alpha*volInfo.ColorTransferFunction[(int)temp*3+2]*256.0;
	    s_remainingOpacity[tempacc]*=(1.0-alpha);
	  }else{
	    pos = s_minmaxTrace[tempacc].y-s_minmaxTrace[tempacc].x;
	  }
	}
	

      }else{ // current position is behind z buffer wall
	
	s_outputVal[tempacc*3]+=(s_remainingOpacity[tempacc])*renInfo.OutputImage[outindex].x;
	s_outputVal[tempacc*3+1]+=(s_remainingOpacity[tempacc])*renInfo.OutputImage[outindex].y;
	s_outputVal[tempacc*3+2]+=(s_remainingOpacity[tempacc])*renInfo.OutputImage[outindex].z;
	
	pos = s_minmaxTrace[tempacc].y-s_minmaxTrace[tempacc].x;
	
      }
                  
    }
    pos++;
    
  }

  //write to output

  renInfo.OutputImage[outindex]=make_uchar4(s_outputVal[tempacc*3], 
                                            s_outputVal[tempacc*3+1], 
					    s_outputVal[tempacc*3+2], 
					    (1-s_remainingOpacity[tempacc])*255.0);
  renInfo.ZBuffer[outindex]=s_zBuffer[tempacc];
}

extern "C"
void CUDArenderAlgo_doRender(const cudaRendererInformation& rendererInfo,
							 const cudaVolumeInformation& volumeInfo)
{
  // setup execution parameters

  dim3 grid(rendererInfo.Resolution[0] / BLOCK_DIM2D, rendererInfo.Resolution[1]/ BLOCK_DIM2D, 1);
  dim3 threads(BLOCK_DIM2D, BLOCK_DIM2D, 1);

  CUT_DEVICE_INIT();

  // execute the kernel
  // Switch to various rendering methods.
  //float transparencyLevel = 1.0;
  
  CUDAkernel_renderAlgo_doIntegrationRender<unsigned char> <<< grid, threads >>>( \
	 rendererInfo,
	 volumeInfo)  
  /*
#define CUDA_KERNEL_CALL(ID, TYPE)   \
	if (inputDataType == ID) \
	 CUDAkernel_renderAlgo_doIntegrationRender<<< grid, threads >>>( \
	 outputData, \
	 colorTransferFunction, \
	 alphaTransferFunction, \
	 zBuffer, \
	 minThreshold, maxThreshold, \
	 sliceDistance, \
	 transparencyLevel)

// Add all the other types.
  CUDA_KERNEL_CALL(VTK_UNSIGNED_CHAR, unsigned char);
  else CUDA_KERNEL_CALL(VTK_CHAR, char);
  else CUDA_KERNEL_CALL(VTK_SHORT, short);
  else CUDA_KERNEL_CALL(VTK_UNSIGNED_SHORT, unsigned short);
  else CUDA_KERNEL_CALL(VTK_FLOAT, float);
  else CUDA_KERNEL_CALL(VTK_DOUBLE, double);
  else CUDA_KERNEL_CALL(VTK_INT, int);
  */


  CUT_CHECK_ERROR("Kernel execution failed");

  return;
}
